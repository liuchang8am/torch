#include "hip/hip_runtime.h"
#include "utils.h"
#include "common.h"

template <typename Dtype>
__global__ void MaxUnpoolForward(const int nthreads, const Dtype* bottom_data, const Dtype* bottom_mask, 
    const int num, const int channels, const int iheight, const int iwidth, const int oheight, const int owidth, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) { //index here indices the input pixels
    int c = (index / iwidth / iheight) % channels;
    int n = index / iwidth / iheight / channels;
    top_data += (n*channels + c)*oheight*owidth;
    int maxind = bottom_mask[index]-1;
    
    top_data[maxind] = bottom_data[index];
  }
}

template <typename Dtype>
__global__ void MaxUnpoolBackward(const int nthreads, const Dtype* top_diff, const Dtype* bottom_mask, 
    const int num, const int channels, const int iheight, const int iwidth, const int oheight, const int owidth, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int c = (index / iwidth / iheight) % channels;
    int n = index / iwidth / iheight / channels;
    top_diff += (n*channels + c)*oheight*owidth; 
    int maxind = bottom_mask[index]-1;

    bottom_diff[index] = top_diff[maxind];
  }
}

static int cunn_SpatialMaxUnpooling_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  int owidth = luaT_getfieldcheckint(L, 1, "owidth");
  int oheight = luaT_getfieldcheckint(L, 1, "oheight");
  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");

  THAssert(THCudaTensor_checkGPU(state, 3, input, output, indices));
  luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  long nInputCols, nInputRows, nInputPlane, batchSize;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  input = THCudaTensor_newContiguous(state, input);
  indices = THCudaTensor_newContiguous(state, indices);
  THCudaTensor_resize4d(state, output, batchSize, nInputPlane, oheight, owidth);
  THCudaTensor_zero(state, output);

  int count = THCudaTensor_nElement(state, input);

  MaxUnpoolForward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, THCudaTensor_data(state, input), THCudaTensor_data(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCudaTensor_data(state, output));

  if(input->nDimension == 3)
    THCudaTensor_resize3d(state, output, nInputPlane, oheight, owidth);

  THCudaTensor_free(state, input);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxUnpooling.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static int cunn_SpatialMaxUnpooling_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  int owidth = luaT_getfieldcheckint(L, 1, "owidth");
  int oheight = luaT_getfieldcheckint(L, 1, "oheight");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");

  THAssert(THCudaTensor_checkGPU(state, 4, input, gradOutput, indices, gradInput));

  long nInputCols, nInputRows, nInputPlane, batchSize;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  input = THCudaTensor_newContiguous(state, input);
  indices = THCudaTensor_newContiguous(state, indices);
  gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  THCudaTensor_resizeAs(state, gradInput, input);
  
  int count = THCudaTensor_nElement(state, input);

  MaxUnpoolBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>> 
      (count, THCudaTensor_data(state, gradOutput), THCudaTensor_data(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCudaTensor_data(state, gradInput));

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxUnpooling.updateGradInput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  // clean
  THCudaTensor_free(state, input);
  THCudaTensor_free(state, gradOutput);

  return 1;
}

static const struct luaL_Reg cunn_SpatialMaxUnpooling__ [] = {
  {"SpatialMaxUnpooling_updateOutput", cunn_SpatialMaxUnpooling_updateOutput},
  {"SpatialMaxUnpooling_updateGradInput", cunn_SpatialMaxUnpooling_updateGradInput},
  {NULL, NULL}
};

void cunn_SpatialMaxUnpooling_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialMaxUnpooling__, "nn");
  lua_pop(L,1);
}
