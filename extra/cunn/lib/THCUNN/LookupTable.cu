#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>

#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

const int WARP_SIZE = 32;

__device__ __forceinline__ bool warpHasCollision(int val)
{
  // Compare our value to the values stored in the next 16 lanes,
  // wrapping around at 32. If any pair of values is the same than
  // there is a collision in the warp.
  bool dup = 0;
  const int laneId = threadIdx.x % 32;

#if __CUDA_ARCH__ >= 300

  #pragma unroll
  for (int i = 1; i <= 16; i++)
  {
    dup |= (__shfl(val, (laneId + i) % 32) == val);
  }

#else

  volatile __shared__ int values[128];
  values[threadIdx.x] = val;
  const int offset = threadIdx.x - laneId;

  #pragma unroll
  for (int i = 1; i <= 16; i++)
  {
    dup |= (values[offset + ((laneId + i) % 32)] == val);
  }

#endif

  return __any(dup) != 0;
}

__global__ void cunn_LookupTable_accGradParametersKernelByFeature(
  float *input, float *gradOutput, float *gradWeight, float scale, long numel, long stride)
{

  const int featureDim = blockIdx.x * 4 + threadIdx.x / 32;
  if (featureDim >= stride)
    return;

  // The strategy here is that each warp handles a single feature
  // dimension.
  // Within that feature dimension, points in the [batch][element]
  // dimension can overlap, and we need to determine if threads want
  // to add to the gradient in a colliding manner.
  // Typically one would use floating-point atomicAdd() to resolve
  // these collisions, but that is non-deterministic if there are
  // collisions. Non-determinism for this code is really bad,
  // especially in RNNs, and is prone to snowballing error.
  // In order to get a deterministic order of execution, we handle
  // non-colliding updates separately from colliding ones. Colliding
  // updates are serialized in their order of execution by using the
  // warp-wide collision detector `warpHasCollision`.
  const int laneId = threadIdx.x % 32;
  for (int i = laneId; i < numel; i += WARP_SIZE)
  {
    int weightIndex = (int) (input[i] - 1);
    float update = gradOutput[i*stride + featureDim] * scale;

    // Check for collision
    if (warpHasCollision(weightIndex))
    {
      // Run all lanes sequentially; warp divergence
      for (int i = 0; i < WARP_SIZE; ++i)
      {
        if (laneId == i)
        {
          gradWeight[weightIndex*stride + featureDim] += update;
        }
      }
    }
    else
    {
      // No collision; warp coherence
      gradWeight[weightIndex*stride + featureDim] += update;
    }
  }
}

__global__ void cunn_LookupTable_accGradParametersKernel(
  float *input,  float *indices, float *gradOutput, float *gradWeight,
  float *count, float defaultScale, long numel, long stride)
{
  int idx = blockIdx.x * 4 + threadIdx.y;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceeding input has the same as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.
  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values proceessed by each thread (grain size)
  const int SZ = 4;

  if (idx < numel && (idx == 0 || input[idx] != input[idx - 1]))
  {
    do
    {
      const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
      const int weightRow = ((int) input[idx] - 1) * stride;
      const int gradOutputRow = ((int) indices[idx] - 1) * stride;
      const float scale = count ? defaultScale / count[idx] : defaultScale;

      float gradient[SZ];
      float weight[SZ];

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++)
      {
        int featureDim = startFeature + ii * WARP_SIZE;
        if (featureDim < stride)
        {
          gradient[ii] = gradOutput[gradOutputRow + featureDim];
          weight[ii] = gradWeight[weightRow + featureDim];
        }
      }

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++)
      {
        weight[ii] += gradient[ii] * scale;
      }

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++)
      {
        int featureDim = startFeature + ii * WARP_SIZE;
        if (featureDim < stride)
        {
          gradWeight[weightRow + featureDim] = weight[ii];
        }
      }

      idx++;
    } while (idx < numel && input[idx] == input[idx - 1]);
  }
}

void THNN_CudaLookupTable_accGradParameters(
  THCState *state,
  THIndexTensor *input,
  THCudaTensor *gradOutput,
  THCudaTensor *gradWeight,
  THIntegerTensor *count,
  THCudaTensor *sorted,
  THCudaTensor *indices,
  bool scaleGradByFreq,
  float scale)
{
  THAssert(THCudaTensor_checkGPU(state, 5, input, gradOutput, gradWeight, sorted, indices));
  if (!(THCudaTensor_isContiguous(state, input) &&
        THCudaTensor_isContiguous(state, gradOutput) &&
        THCudaTensor_isContiguous(state, gradWeight)))
  {
    THError("Tensors must be contiguous");
  }

  int nDim = THCudaTensor_nDimension(state, input);
  if (nDim != 1 && nDim != 2)
    THError("input must be a vector or matrix");

  long numel = THCudaTensor_nElement(state, input);
  long stride = gradWeight->stride[0];

  hipStream_t stream = THCState_getCurrentStream(state);

  if (numel <= 768 && !scaleGradByFreq)
  {
    cunn_LookupTable_accGradParametersKernelByFeature<<<DIVUP(stride,4), 128, 0, stream>>>(
      THCudaTensor_data(state, input),
      THCudaTensor_data(state, gradOutput),
      THCudaTensor_data(state, gradWeight),
      scale,
      numel,
      stride
    );

    return;
  }

  THCudaTensor_resizeAs(state, sorted, input);
  THCudaTensor_resizeAs(state, indices, input);

  // Sort the inputs into sorted with the corresponding indices
  THCudaTensor_sort(state, sorted, indices, input, 0, 0);

  float *sorted_data = THCudaTensor_data(state, sorted);
  float *indices_data = THCudaTensor_data(state, indices);
  float *count_data = NULL;

  if (scaleGradByFreq)
  {
    THIntegerTensor_(resizeAs)(state, count, input);
    count_data = THIntegerTensor_(data)(state, count);

    thrust::device_ptr<float> sorted_ptr(sorted_data);
    thrust::device_ptr<float> count_ptr(count_data);

    // Compute an increasing sequence per unique item in sorted:
    // sorted: 2 5 5 5 7 7 8 9 9
    //  count: 1 1 2 3 1 2 1 1 2
    thrust::inclusive_scan_by_key(
      sorted_ptr,
      sorted_ptr + numel,
      thrust::make_constant_iterator(1),
      count_ptr
    );

    // Take the maximum of each count per unique key in reverse:
    // sorted: 2 5 5 5 7 7 8 9 9
    //  count: 1 3 3 3 2 2 1 2 2
    thrust::inclusive_scan_by_key(
      thrust::make_reverse_iterator(sorted_ptr + numel),
      thrust::make_reverse_iterator(sorted_ptr),
      thrust::make_reverse_iterator(count_ptr + numel),
      thrust::make_reverse_iterator(count_ptr + numel),
      thrust::equal_to<float>(),
      thrust::maximum<float>()
    );
  }

  dim3 grid(DIVUP(numel,4), DIVUP(stride,128));
  dim3 block(32, 4);
  cunn_LookupTable_accGradParametersKernel<<<grid, block, 0, stream>>>(
    sorted_data,
    indices_data,
    THCudaTensor_data(state, gradOutput),
    THCudaTensor_data(state, gradWeight),
    count_data,
    scale,
    numel,
    stride
  );
}
