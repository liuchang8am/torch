#include "hip/hip_runtime.h"
#include "THCUNN.h"

struct sigmoidupdateOutput_functor
{
  __device__ void operator()(float *output, const float *input) const
  {
    *output = 1./(1.+ exp(-*input));
  }
};

void THNN_CudaSigmoid_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output)
{
  THAssert(THCudaTensor_checkGPU(state, 2, input, output));
  THCudaTensor_resizeAs(state, output, input);
  THCudaTensor_pointwiseApply2(state, output, input, sigmoidupdateOutput_functor());
}

struct sigmoidupdateGradInput_functor
{
  __device__ void operator()(float *gradInput, const float *output, const float *gradOutput) const
  {
    *gradInput = *gradOutput * (1.-*output) * (*output);
  }
};

void THNN_CudaSigmoid_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput, THCudaTensor *gradInput, THCudaTensor *output)
{
  THAssert(THCudaTensor_checkGPU(state, 3, output, gradOutput, gradInput));
  THCudaTensor_resizeAs(state, gradInput, output);
  THCudaTensor_pointwiseApply3(state, gradInput, output, gradOutput, sigmoidupdateGradInput_functor());
}
