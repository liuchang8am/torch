#include "hip/hip_runtime.h"
#include "THCUNN.h"

struct tanhupdateOutput_functor
{
  __device__ void operator()(float *output, const float *input) const
  {
    *output = tanh(*input);
  }
};

void THNN_CudaTanh_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output)
{
  THAssert(THCudaTensor_checkGPU(state, 2, input, output));
  THCudaTensor_resizeAs(state, output, input);
  THCudaTensor_pointwiseApply2(state, output, input, tanhupdateOutput_functor());
}

struct tanhupdateGradInput_functor
{
  __device__ void operator()(float *gradInput, const float *output, const float *gradOutput) const
  {
    *gradInput = *gradOutput * (1 - *output * *output);
  }
};

void THNN_CudaTanh_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput, THCudaTensor *gradInput, THCudaTensor *output)
{
  THAssert(THCudaTensor_checkGPU(state, 3, output, gradOutput, gradInput));
  THCudaTensor_resizeAs(state, gradInput, output);
  THCudaTensor_pointwiseApply3(state, gradInput, output, gradOutput, tanhupdateGradInput_functor());
}
