#include "THCApply.cuh"

inline int curGPU() {
  int curDev;
  THCudaCheck(hipGetDevice(&curDev));
  return curDev;
}

#include "generic/THCTensorCopy.cu"
#include "THCGenerateAllTypes.h"
