#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCBlas.h"
#include "THCTensorCopy.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"

struct TensorAddConstantOp {
  TensorAddConstantOp(float v) : val(v) {}
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = *in + val;
  }

  __device__ __forceinline__ void operator()(float* v) {
    *v += val;
  }

  const float val;
};

void THCudaTensor_add(THCState *state, THCudaTensor *self_, THCudaTensor *src_, float value)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THCudaTensor_pointwiseApply1(state, self_, TensorAddConstantOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src_);

    if (!THCudaTensor_pointwiseApply2(state, self_, src_, TensorAddConstantOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorMulConstantOp {
  TensorMulConstantOp(float v) : val(v) {}
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = *in * val;
  }

  __device__ __forceinline__ void operator()(float* v) {
    *v *= val;
  }

  const float val;
};

void THCudaTensor_mul(THCState *state, THCudaTensor *self_, THCudaTensor *src_, float value)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THCudaTensor_pointwiseApply1(state, self_, TensorMulConstantOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src_);

    if (!THCudaTensor_pointwiseApply2(state, self_, src_, TensorMulConstantOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

void THCudaTensor_div(THCState* state, THCudaTensor *self_, THCudaTensor *src_, float value)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(value != 0.0f, 3, "divide by zero");

  if (self_ == src_) {
    if (!THCudaTensor_pointwiseApply1(state, self_, TensorMulConstantOp(1.0f / value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src_);

    if (!THCudaTensor_pointwiseApply2(state, self_, src_, TensorMulConstantOp(1.0f / value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

template <int Upper>
struct TensorTriOp {
  TensorTriOp(float *start_, long stride0_, long stride1_, long k_)
    : start(start_), stride0(stride0_), stride1(stride1_), k(k_) {}

  __device__ __forceinline__ int mask(float *in) {
    ptrdiff_t n = in - start;
    long row, col;
    if (stride0 > stride1)
    {
      row = (long) (n / stride0);
      col = (long) ((n % stride0) / stride1);
    }
    else
    {
      row = (long) ((n % stride1) / stride0);
      col = (long) (n / stride1);
    }

    return Upper ? (col - row >= k) : (col - row <= k);
  }

  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = mask(in) ? *in : 0;
  }

  __device__ __forceinline__ void operator()(float* v) {
    if (!mask(v))
      *v = 0;
  }

  const float *start;
  const long stride0, stride1, k;
};

void THCudaTensor_tril(THCState *state, THCudaTensor *self_, THCudaTensor *src_, long k)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THCudaTensor *src = src_;
  if (self_ == src_)
    src = THCudaTensor_newContiguous(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  float *start = THCudaTensor_data(state, src) + src->storageOffset;

  TensorTriOp<0> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THCudaTensor_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src);

    if (!THCudaTensor_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THCudaTensor_freeCopyTo(state, src, src_);

  THCudaCheck(hipGetLastError());
}

void THCudaTensor_triu(THCState *state, THCudaTensor *self_, THCudaTensor *src_, long k)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THCudaTensor *src = src_;
  if (self_ == src_)
    src = THCudaTensor_newContiguous(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  float *start = THCudaTensor_data(state, src) + src->storageOffset;

  TensorTriOp<1> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THCudaTensor_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src);

    if (!THCudaTensor_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THCudaTensor_freeCopyTo(state, src, src_);

  THCudaCheck(hipGetLastError());
}
