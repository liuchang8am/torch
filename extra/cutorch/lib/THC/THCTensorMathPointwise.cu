#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCBlas.h"
#include "THCTensorCopy.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC)                   \
  struct Tensor##NAME##Op {                                             \
    __device__ __forceinline__ void operator()(float* out, float* in) const { \
      *out = CFUNC(*in);                                                \
    }                                                                   \
                                                                        \
    __device__ __forceinline__ void operator()(float* v) const {        \
      *v = CFUNC(*v);                                                   \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCudaTensor_##NAME(THCState* state, THCudaTensor* self_, THCudaTensor* src) { \
    THAssert(THCudaTensor_checkGPU(state, 2, self_, src));                \
    if (self_ == src) {                                                 \
      if (!THCudaTensor_pointwiseApply1(state, self_, Tensor##NAME##Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING); \
      }                                                                 \
    } else {                                                            \
      THCudaTensor_resizeAs(state, self_, src);                         \
                                                                        \
      if (!THCudaTensor_pointwiseApply2(state, self_, src, Tensor##NAME##Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING); \
      }                                                                 \
    }                                                                   \
                                                                        \
    THCudaCheck(hipGetLastError());                                    \
  }

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log, log)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log1p, log1p)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(exp, exp)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(cos, cos)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(acos, acos)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(cosh, cosh)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(sin, sin)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(asin, asin)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(sinh, sinh)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(tan, tan)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(atan, atan)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(tanh, tanh)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(sqrt, sqrt)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(ceil, ceil)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(floor, floor)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(abs, fabs)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(round, roundf)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(neg, -)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(cinv, 1.0f / )

#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC

struct TensorSigmoidOp {
  __device__ __forceinline__ void operator()(float* out, float* in) const {
    *out = 1.0f / (1.0f + expf(- *in));
  }

  __device__ __forceinline__ void operator()(float* v) const {
    *v = 1.0f / (1.0f + expf(- *v));
  }
};

void THCudaTensor_sigmoid(THCState* state, THCudaTensor* self_, THCudaTensor* src) {
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src));
  if (self_ == src) {
    if (!THCudaTensor_pointwiseApply1(state, self_, TensorSigmoidOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src);

    if (!THCudaTensor_pointwiseApply2(state, self_, src, TensorSigmoidOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorAddOp {
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out += *in;
  }

  __device__ __forceinline__ void operator()(float* out, float* in1, float* in2) {
    *out = *in1 + *in2;
  }
};

struct TensorCAddOp {
  TensorCAddOp(float v) : val(v) {}

  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out += val * *in;
  }

  __device__ __forceinline__ void operator()(float* out, float* in1, float* in2) {
    *out = *in1 + val * *in2;
  }

  float val;
};

void THCudaTensor_cadd(THCState *state, THCudaTensor *self_, THCudaTensor* src1, float value, THCudaTensor *src2)
{
  THAssert(THCudaTensor_checkGPU(state, 3, self_, src1, src2));
  THArgCheck(THCudaTensor_nElement(state, src1) ==
             THCudaTensor_nElement(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    if (value == 1.0f) {
      // self += src2
      if (!THCudaTensor_pointwiseApply2(state, self_, src2, TensorAddOp())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self += value * src2
      if (!THCudaTensor_pointwiseApply2(state, self_, src2, TensorCAddOp(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src1);

    if (value == 1.0f) {
      // self = src1 + src2
      if (!THCudaTensor_pointwiseApply3(state, self_, src1, src2, TensorAddOp())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self = src1 + value * src2
      if (!THCudaTensor_pointwiseApply3(state, self_, src1, src2, TensorCAddOp(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorMulOp {
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out *= *in;
  }

  __device__ __forceinline__ void operator()(float* out, float* in1, float* in2) {
    *out = *in1 * *in2;
  }
};

void THCudaTensor_cmul(THCState *state, THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THAssert(THCudaTensor_checkGPU(state, 3, self_, src1, src2));
  THArgCheck(THCudaTensor_nElement(state, src1) ==
             THCudaTensor_nElement(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self *= src2
    if (!THCudaTensor_pointwiseApply2(state, self_, src2, TensorMulOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src1);

    // self = src1 * src2
    if (!THCudaTensor_pointwiseApply3(state, self_, src1, src2, TensorMulOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorMaxOp {
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = max(*out, *in);
  }

  __device__ __forceinline__ void operator()(float* out, float* in1, float* in2) {
    *out = max(*in1, *in2);
  }
};

void THCudaTensor_cmax(THCState *state, THCudaTensor *self, THCudaTensor *src1, THCudaTensor *src2)
{
  THAssert(THCudaTensor_checkGPU(state, 3, self, src1, src2));
  THArgCheck(THCudaTensor_nElement(state, src1) ==
             THCudaTensor_nElement(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THCudaTensor_pointwiseApply2(state, self, src2, TensorMaxOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self, src1);
    if (!THCudaTensor_pointwiseApply3(state, self, src1, src2, TensorMaxOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

struct TensorMinOp {
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = min(*out, *in);
  }

  __device__ __forceinline__ void operator()(float* out, float* in1, float* in2) {
    *out = min(*in1, *in2);
  }
};

void THCudaTensor_cmin(THCState *state, THCudaTensor *self, THCudaTensor *src1, THCudaTensor *src2)
{
  THAssert(THCudaTensor_checkGPU(state, 3, self, src1, src2));
  THArgCheck(THCudaTensor_nElement(state, src1) ==
             THCudaTensor_nElement(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THCudaTensor_pointwiseApply2(state, self, src2, TensorMinOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self, src1);
    if (!THCudaTensor_pointwiseApply3(state, self, src1, src2, TensorMinOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

struct TensorMaxValueOp {
  TensorMaxValueOp(float v) : val(v) {}

  __device__ __forceinline__ void operator()(float* out) {
    *out = max(*out, val);
  }

  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = max(*in, val);
  }

  float val;
};

void THCudaTensor_cmaxValue(THCState *state, THCudaTensor *self, THCudaTensor *src, float value)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self, src));

  if (self == src) {
    if (!THCudaTensor_pointwiseApply1(state, self, TensorMaxValueOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self, src);
    if (!THCudaTensor_pointwiseApply2(state, self, src, TensorMaxValueOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

struct TensorMinValueOp {
  TensorMinValueOp(float v) : val(v) {}

  __device__ __forceinline__ void operator()(float* out) {
    *out = min(*out, val);
  }

  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = min(*in, val);
  }

  float val;
};

void THCudaTensor_cminValue(THCState *state, THCudaTensor *self, THCudaTensor *src, float value)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self, src));

  if (self == src) {
    if (!THCudaTensor_pointwiseApply1(state, self, TensorMinValueOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self, src);
    if (!THCudaTensor_pointwiseApply2(state, self, src, TensorMinValueOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}
